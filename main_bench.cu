#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>

#include <cuda/std/chrono>
#include <nvbench/nvbench.cuh>
#include <random>

#include "topk.cuh"

constexpr int N = 2e7;

struct GPU_RNG {
  thrust::random::taus88 rng;
  thrust::random::uniform_real_distribution<float> dist;
  GPU_RNG() : rng(std::random_device{}()), dist(0.f, 1.f) {}
  __device__ float operator()() { return dist(rng); }
} rng;

void generate_random_device_vector(thrust::device_vector<float> &v) {
  thrust::generate(thrust::device, v.begin(), v.end(), rng);
}

struct TopK_Benchmark {
  using test_func_t = void (*)(float *, int, int, float *, hipStream_t);
  test_func_t p;
  TopK_Benchmark(test_func_t p) : p(p) {}

  void operator()(nvbench::state &state) {
    const auto k = state.get_int64("k");
    auto src_vec = thrust::device_vector<float>(N);
    auto result = thrust::device_vector<float>(k);
    // state.add_global_memory_reads<nvbench::int32_t>(N, "DataSize");
    state.exec(
        nvbench::exec_tag::timer,
        [k, &src_vec, &result, this](nvbench::launch &launch, auto &timer) {
          generate_random_device_vector(src_vec);
          timer.start();
          this->p(thrust::raw_pointer_cast(src_vec.data()), N, k,
                  thrust::raw_pointer_cast(result.data()), launch.get_stream());
          timer.stop();
#if 0
      thrust::sort(src_vec.begin(), src_vec.end(), thrust::greater<float>());
      thrust::sort(result.begin(), result.end(), thrust::greater<float>());
      for (int i = 0; i < k; i++) {
        if (result[i] != src_vec[i]) {
          printf("i = %d: %f != %f\n", i, float(result[i]), float(src_vec[i]));
        }
      }
#endif
        });
  }
};

struct TopK_Benchmark_WithBuffer {
  using test_func_t = void (*)(float *, int, int, float *, hipStream_t,
                               float *);
  test_func_t p;
  TopK_Benchmark_WithBuffer(test_func_t p) : p(p) {}

  void operator()(nvbench::state &state) {
    const auto k = state.get_int64("k");
    auto src_vec = thrust::device_vector<float>(N);
    auto buffer = thrust::device_vector<float>(N);
    auto result = thrust::device_vector<float>(k);
    // state.add_global_memory_reads<nvbench::int32_t>(N, "DataSize");
    state.exec(nvbench::exec_tag::timer | nvbench::exec_tag::sync,
               [k, &src_vec, &result, &buffer, this](nvbench::launch &launch,
                                                     auto &timer) {
                 generate_random_device_vector(src_vec);
                 timer.start();
                 this->p(thrust::raw_pointer_cast(src_vec.data()), N, k,
                         thrust::raw_pointer_cast(result.data()),
                         launch.get_stream(),
                         thrust::raw_pointer_cast(buffer.data()));
                 timer.stop();
#if 0
      thrust::sort(src_vec.begin(), src_vec.end(), thrust::greater<float>());
      thrust::sort(result.begin(), result.end(), thrust::greater<float>());
      for (int i = 0; i < k; i++) {
        if (result[i] != src_vec[i]) {
          printf("i = %d: %f != %f\n", i, float(result[i]), float(src_vec[i]));
        }
      }
#endif
               });
  }
};

auto yxz_topk_benchmark = TopK_Benchmark(yxz_topk);
auto yxz_topk_heap_benchmark = TopK_Benchmark(yxz_topk_heap);
auto anil_bitonic_benchmark = TopK_Benchmark_WithBuffer(anil_bitonic<float>);

constexpr int range_max = 48;

NVBENCH_BENCH(yxz_topk_benchmark)
    .add_int64_axis("k", nvbench::range(1, range_max));
NVBENCH_BENCH(yxz_topk_heap_benchmark)
    .add_int64_axis("k", nvbench::range(1, range_max));
NVBENCH_BENCH(anil_bitonic_benchmark)
    .add_int64_axis("k", nvbench::range(1, range_max));
